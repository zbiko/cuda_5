#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <thread>

// Kernel to simulate a 3-second sleep
__global__ void sleepKernel() {
    // Sleep for ~3 seconds (busy wait loop)
    unsigned long long startClock = clock64();
    unsigned long long waitClock = 9000000000; // 3 seconds in nanoseconds for a busy GPU clock wait
    while (clock64() - startClock < waitClock) {
        // Busy waiting
    }
}

// Kernel to perform a quick task
__global__ void quickKernel() {
    printf("Quick kernel executed!\n");
}

int main() {
    // Initialize CUDA Driver API
    hipInit(0);

    // Get the first CUDA device
    hipDevice_t device;
    hipDeviceGet(&device, 0);

    // Create a CUDA context
    hipCtx_t ctx;
    hipCtxCreate(&ctx, 0, device);

    hipCtxSetCurrent(ctx);

    // Create two CUDA streams with non-blocking behavior
    hipStream_t stream1, stream2;
    hipStreamCreateWithFlags(&stream1, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&stream2, hipStreamNonBlocking);

    // Create CUDA events to measure timing
    hipEvent_t start, event1, event2;
    hipEventCreateWithFlags(&start, hipEventDefault);
    hipEventCreateWithFlags(&event1, hipEventDefault);
    hipEventCreateWithFlags(&event2, hipEventDefault);

    // Record the start event on the default stream
    hipEventRecord(start, 0);

    // Launch the sleep kernel on stream1
    std::cout << "Launching sleep kernel..." << std::endl;
    sleepKernel<<<1, 1, 0, stream1>>>();
    hipEventRecord(event1, stream1);

    // Launch the quick kernel on stream2
    std::cout << "Launching quick kernel..." << std::endl;
    quickKernel<<<1, 1, 0, stream2>>>();
    hipEventRecord(event2, stream2);

    hipStreamSynchronize(stream1); // Synchronize only stream1
    hipStreamSynchronize(stream2); // Synchronize only stream2
    // Measure elapsed time
    float elapsed1, elapsed2;
    hipEventElapsedTime(&elapsed1, start, event1);
    hipEventElapsedTime(&elapsed2, start, event2);

    std::cout << "Sleep kernel finished at: " << elapsed1 << " ms" << std::endl;
    std::cout << "Quick kernel finished at: " << elapsed2 << " ms" << std::endl;

    // Cleanup
    hipEventDestroy(start);
    hipEventDestroy(event1);
    hipEventDestroy(event2);
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    hipCtxDestroy(ctx);

    return 0;
}
